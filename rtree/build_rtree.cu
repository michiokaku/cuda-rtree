#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "build_rtree.cuh"
#include "readObj.h"
#include "dataStruct.h"

#include <stdio.h>
#include <stdlib.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include "sort.cuh"
#include  <direct.h>

#define GETVERTEX(o,f,j) o.vertexArray[f.v[j]];
#define THREAD_PER_BLOCK 256
#define SCALE 0.1 //scale the object;

int GetBlockCount(int threadCount) {

	int blockCount = threadCount / THREAD_PER_BLOCK;

	if ((threadCount%THREAD_PER_BLOCK) != 0)
	{
		++blockCount;
	}

	return blockCount;
}

__global__ void scaleObj(fVertex *fv,int length)
{
	int tid = blockIdx.x *blockDim.x + threadIdx.x;

	if (tid < length)
	{
		fVertex f = fv[tid];
		f.x *= SCALE;
		f.y *= SCALE;
		f.z *= SCALE;

		fv[tid] = f;
	}
}

__global__ void buildBoxKernel(box *b, obj o, fVertex *midpoint)
{
	int tid = blockIdx.x *blockDim.x + threadIdx.x;

	if (tid < o.faceCount)
	{

		//get vertex from vertex
		face f = o.faceArray[tid];
		fVertex v1 = GETVERTEX(o, f, 0);
		fVertex v2 = GETVERTEX(o, f, 1);
		fVertex v3 = GETVERTEX(o, f, 2);

		//get max and min value of x,y,z
		float xMax = v1.x;
		xMax = fmaxf(v2.x, xMax);
		xMax = fmaxf(v3.x, xMax);

		float xMin = v1.x;
		xMin = fminf(v2.x, xMin);
		xMin = fminf(v3.x, xMin);

		float yMax = v1.y;
		yMax = fmaxf(v2.y, yMax);
		yMax = fmaxf(v3.y, yMax);

		float yMin = v1.y;
		yMin = fminf(v2.y, yMin);
		yMin = fminf(v3.y, yMin);

		float zMax = v1.z;
		zMax = fmaxf(v2.z, zMax);
		zMax = fmaxf(v3.z, zMax);

		float zMin = v1.z;
		zMin = fminf(v2.z, zMin);
		zMin = fminf(v3.z, zMin);

		//build box on the register
		box regBox;
		regBox.xMax = xMax;
		regBox.xMin = xMin;

		regBox.yMax = yMax;
		regBox.yMin = yMin;

		regBox.zMax = zMax;
		regBox.zMin = zMin;


		//return regBox
		b[tid] = regBox;

		//count minpoint
		fVertex mp;
		mp.x = (xMax + xMin) / 2.0f;
		mp.y = (yMax + yMin) / 2.0f;
		mp.z = (zMax + zMin) / 2.0f;

		//return midpoint
		midpoint[tid] = mp;
	}
}

void buildBox(box *b, obj o, fVertex *midpoint)
{
	scaleObj << <GetBlockCount(o.vertexCount), THREAD_PER_BLOCK >> > (o.vertexArray, o.vertexCount);
	buildBoxKernel << <GetBlockCount(o.faceCount), THREAD_PER_BLOCK >> > (b, o, midpoint);

	//box *host_b = (box*)malloc(o.faceCount * sizeof(box));
	//hipMemcpy(host_b, b, o.vertexCount * sizeof(fVertex), hipMemcpyDeviceToHost);
}

__device__ unsigned int expandBits(unsigned int v)
{
	v = (v * 0x00010001u) & 0xFF0000FFu;
	v = (v * 0x00000101u) & 0x0F00F00Fu;
	v = (v * 0x00000011u) & 0xC30C30C3u;
	v = (v * 0x00000005u) & 0x49249249u;
	return v;
}

// Calculates a 30-bit Morton code for the
// given 3D point located within the unit cube [0,1].
__device__ unsigned int morton3D(float x, float y, float z)
{
	x = min(max(x * 1024.0f, 0.0f), 1023.0f);
	y = min(max(y * 1024.0f, 0.0f), 1023.0f);
	z = min(max(z * 1024.0f, 0.0f), 1023.0f);
	unsigned int xx = expandBits((unsigned int)x);
	unsigned int yy = expandBits((unsigned int)y);
	unsigned int zz = expandBits((unsigned int)z);
	return xx * 4 + yy * 2 + zz;
}

__global__ void CountZorder(unsigned int * zOrder, fVertex * point, int length)
{
	int tid = blockIdx.x *blockDim.x + threadIdx.x;

	if (tid < length)
	{
		fVertex p = point[tid];
		zOrder[tid] = morton3D(p.x, p.y, p.z);
	}
}

__global__ void test(unsigned int * zOrder)
{
	int tid = blockIdx.x *blockDim.x + threadIdx.x;
}

void SortBox(box *b, fVertex *midpoint, int length)
{
	//build zOrder value of the midpoint
	unsigned int * zOrder;
	hipMalloc((void**)&zOrder, length * sizeof(unsigned int));

	CountZorder << <GetBlockCount(length), THREAD_PER_BLOCK >> > (zOrder, midpoint, length);

	b = sort_cub(zOrder, b, length);
}

__device__ box BigerBox(box a, box b)
{
	a.xMax = fmaxf(a.xMax, b.xMax);
	a.xMin = fmaxf(a.xMin, b.xMin);

	a.yMax = fmaxf(a.yMax, b.yMax);
	a.yMin = fmaxf(a.yMin, b.yMin);

	a.zMax = fmaxf(a.zMax, b.zMax);
	a.zMin = fmaxf(a.zMin, b.zMin);
}

__global__ void FirstMergeBoxKernel(node *n ,box *b,int offset,int parentLength,int length)
{
	int tid = blockIdx.x *blockDim.x + threadIdx.x;

	if (tid < parentLength)
	{
		int childIndex = tid * CHILD_COUNT;
		box rb = b[childIndex];//the box of node
		node rn;
		rn.child[0] = childIndex;

		//count the number of children
		int childNum = CHILD_COUNT;
		if (tid == parentLength - 1)childNum = length%CHILD_COUNT;

		box cb;
		for (int i = 1;i < childNum;i++)
		{
			rn.child[i] = childIndex + i;
			cb = b[childIndex + i];
			rb = BigerBox(rb, cb);
		}

		//return node
		rn.b = rb;
		n[offset + tid] = rn;
	}
}

rtree mergeBox(box *b,int length)
{
	rtree r;

	r.nodeCount = 0;
	r.layer = 0;
	//Do not count the last layer

	int len = length;
	do
	{
		len = (len + CHILD_COUNT - 1) / CHILD_COUNT;
		r.nodeCount += len;
		r.layer++;
	} 
	while (len > 1);

	hipMalloc((void**)&r.n, r.nodeCount * sizeof(unsigned int));

	len = length;
	len = (len + CHILD_COUNT - 1) / CHILD_COUNT;
	int offset = r.nodeCount - len;
	//FirstMergeBoxKernel << <GetBlockCount(len), THREAD_PER_BLOCK >> > (r.n, b, offset, len,length);

	for (int i = 0;i < r.layer;i++)
	{

	}

	return r;
}

void buildRtree()
{
	//read obj
	obj o = ReadObj("C:\\Users\\chenxiyu\\Documents\\Visual Studio 2015\\Projects\\rtree\\media\\dragon.obj");

	//build obj on the device
	obj dev_o;
	dev_o.faceCount = o.faceCount;
	dev_o.vertexCount = o.vertexCount;
	hipMalloc((void**)&dev_o.vertexArray, dev_o.vertexCount * sizeof(fVertex));
	hipMalloc((void**)&dev_o.faceArray, dev_o.faceCount * sizeof(face));

	//copy the data of obj witch from host to device
	hipMemcpy(dev_o.vertexArray, o.vertexArray, dev_o.vertexCount * sizeof(fVertex), hipMemcpyHostToDevice);
	hipMemcpy(dev_o.faceArray, o.faceArray, dev_o.vertexCount * sizeof(face), hipMemcpyHostToDevice);

	//build box for each face
	fVertex *dev_midpoint;      //build the midpoint of each box
	box *dev_box;               //build the box on the device
	hipMalloc((void**)&dev_midpoint, dev_o.faceCount * sizeof(fVertex));
	hipMalloc((void**)&dev_box, o.faceCount * sizeof(box));
	buildBox(dev_box, dev_o, dev_midpoint);

	//sort box by the zorder
	SortBox(dev_box, dev_midpoint, o.faceCount);

	//mergeBox(dev_box, o.faceCount);

	//hipFree(dev_midpoint);
}