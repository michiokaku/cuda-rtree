#include "sort.cuh"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <algorithm>


box * sort_cub(unsigned int * key, box * value,int length)
{
	unsigned int *key_out;
	box *value_out;

	hipMalloc((void**)&key_out, length * sizeof(unsigned int));
	hipMalloc((void**)&value_out, length * sizeof(box));
	// Determine temporary device storage requirements
	void     *d_temp_storage = NULL;
	size_t   temp_storage_bytes = 0;

	hipcub::DeviceRadixSort::SortPairs<unsigned int, box>(d_temp_storage, temp_storage_bytes,
		key, key_out, value, value_out, length);

	// Allocate temporary storage
	hipMalloc(&d_temp_storage, temp_storage_bytes);

	// Run sorting operation
	hipcub::DeviceRadixSort::SortPairs<unsigned int, box>(d_temp_storage, temp_storage_bytes,
		key, key_out, value, value_out, length);

	hipFree(key);
	hipFree(value);
	hipFree(key_out);

	hipFree(d_temp_storage);

	return value_out;
}